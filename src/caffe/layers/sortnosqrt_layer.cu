#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/sortnosqrt_layer.hpp"
#include "caffe/util/math_functions.hpp"



namespace caffe {

template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out
    ) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : 0;
  }
}

template <typename Dtype>
void SortnosqrtLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_data_1 = bottom[1]->gpu_data();
  Dtype* bottom_relu_data = bottom_relu.mutable_gpu_data();
  Dtype* bottom_relu_data_1 = bottom_relu_1.mutable_gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();

  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom_relu_data);
  
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data_1, bottom_relu_data_1);
  
  caffe_gpu_mul(count, bottom_relu.gpu_data(), bottom_relu_1.gpu_data(), after_prod.mutable_gpu_data());
  caffe_gpu_add(count, bottom_data, bottom_data_1, top_data);
  caffe_gpu_add(count, top[0]->gpu_data(), after_prod.gpu_data(), top_data);
}

template <typename Dtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, const Dtype* in_data_1, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * (1.0 + in_data_1[index]*(in_data[index] > 0));
  }
}

template <typename Dtype>
void SortnosqrtLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_data_1 = bottom[1]->gpu_data();

    Dtype* bottom_relu_data = bottom_relu.mutable_gpu_data();
    Dtype* bottom_relu_data_1 = bottom_relu_1.mutable_gpu_data();

    const Dtype* top_diff = top[0]->gpu_diff();

    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    Dtype* bottom_diff_1 = bottom[1]->mutable_gpu_diff();

    const int count = bottom[0]->count();
    ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, bottom_relu_data);
    ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data_1, bottom_relu_data_1);

    ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_relu_data_1, bottom_diff);    
    ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data_1, bottom_relu_data, bottom_diff_1);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(SortnosqrtLayer);

}  // namespace caffe
